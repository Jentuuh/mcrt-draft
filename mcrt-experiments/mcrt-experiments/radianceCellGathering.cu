#include "hip/hip_runtime.h"
#pragma once

#include <optix_device.h>
#include "random.hpp"
#include "vec_math.hpp"

#include "LaunchParams.hpp"
#include "glm/glm.hpp"

#define PI 3.14159265358979323846f
#define EPSILON 0.0000000000002f

using namespace mcrt;

namespace mcrt {

    extern "C" __constant__ LaunchParamsRadianceCellGather optixLaunchParams;

    static __forceinline__ __device__
        void* unpackPointer(uint32_t i0, uint32_t i1)
    {
        const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
        void* ptr = reinterpret_cast<void*>(uptr);
        return ptr;
    }

    static __forceinline__ __device__
        void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    static __forceinline__ __device__ T* getPRD()
    {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T*>(unpackPointer(u0, u1));
    }


    extern "C" __global__ void __closesthit__radiance__cell__gathering()
    {
    }

    extern "C" __global__ void __anyhit__radiance__cell__gathering() {
        // Do nothing
        printf("Hit!HEHEE");
    }

    extern "C" __global__ void __miss__radiance__cell__gathering()
    {

    }

    extern "C" __global__ void __raygen__renderFrame__cell__gathering()
    {
        // Get thread indices
        const int uIndex = optixGetLaunchIndex().x;
        const int vIndex = optixGetLaunchIndex().y;

        // TODO: SKIP PIXELS THAT ARE BLACK!
        uint32_t lightSrcColor = optixLaunchParams.lightSourceTexture.colorBuffer[vIndex * optixLaunchParams.lightSourceTexture.size + uIndex];
        //printf("%d", lightSrcColor);

        glm::vec3 UVWorldPos = optixLaunchParams.uvWorldPositions.UVDataBuffer[vIndex * optixLaunchParams.lightSourceTexture.size + uIndex].worldPosition;
        const glm::vec3 UVNormal = optixLaunchParams.uvWorldPositions.UVDataBuffer[vIndex * optixLaunchParams.lightSourceTexture.size + uIndex].worldNormal;
        // We apply a small offset of 0.00001f in the direction of the normal to the UV world pos, to 'mitigate' floating point rounding errors causing false occlusions/illuminations
        UVWorldPos = glm::vec3{ UVWorldPos.x + UVNormal.x * 0.00001f, UVWorldPos.y + UVNormal.y * 0.00001f, UVWorldPos.z + UVNormal.z * 0.00001f };
        
        // Iterate over all non-empty cells
        for (int i = 0; i < optixLaunchParams.nonEmptyCells.size; i++)
        {
            glm::vec3 cellCenter = optixLaunchParams.nonEmptyCells.centers[i];
            glm::vec3 lightToCellDir = { cellCenter.x - UVWorldPos.x, cellCenter.y - UVWorldPos.y, cellCenter.z - UVWorldPos.z };

            float3 rayOrigin3f = float3{ UVWorldPos.x, UVWorldPos.y, UVWorldPos.z };
            float3 rayDir3f = float3{ lightToCellDir.x, lightToCellDir.y, lightToCellDir.z };

            optixTrace(optixLaunchParams.traversable,
                rayOrigin3f,
                rayDir3f,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_NONE,
                0,  // SBT offset
                1,  // SBT stride
                0  // missSBTIndex 
                );
        }
    }
}