#include "hip/hip_runtime.h"
#pragma once

#include <optix_device.h>
#include "random.hpp"
#include "vec_math.hpp"

#include "LaunchParams.hpp"
#include "glm/glm.hpp"

#include "spherical_harmonics.cuh"

#define PI 3.14159265358979323846f
#define EPSILON 0.0000000000002f
#define NUM_SAMPLES_PER_STRATIFY_CELL 5

using namespace mcrt;


namespace mcrt {
    extern "C" __constant__ LaunchParamsRadianceCellScatter optixLaunchParams;

    static __forceinline__ __device__ RadianceCellScatterPRD loadRadianceCellScatterPRD()
    {
        RadianceCellScatterPRD prd = {};

        prd.distanceToClosestIntersection = __uint_as_float(optixGetPayload_0());
        prd.rayOrigin.x = __uint_as_float(optixGetPayload_1());
        prd.rayOrigin.y = __uint_as_float(optixGetPayload_2());
        prd.rayOrigin.z = __uint_as_float(optixGetPayload_3());

        return prd;
    }

    static __forceinline__ __device__ void storeRadianceCellScatterPRD(RadianceCellScatterPRD prd)
    {
        optixSetPayload_0(__float_as_uint(prd.distanceToClosestIntersection));
        optixSetPayload_1(__float_as_uint(prd.rayOrigin.x));
        optixSetPayload_2(__float_as_uint(prd.rayOrigin.y));
        optixSetPayload_3(__float_as_uint(prd.rayOrigin.z));
    }


    extern "C" __global__ void __closesthit__radiance__cell__scattering__scene()
    {
        const MeshSBTDataRadianceCellScatter& sbtData
            = *(const MeshSBTDataRadianceCellScatter*)optixGetSbtDataPointer();

        const int primID = optixGetPrimitiveIndex();
        const glm::ivec3 index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        const glm::vec3 intersectionWorldPos =
            (1.f - u - v) * sbtData.vertex[index.x]
            + u * sbtData.vertex[index.y]
            + v * sbtData.vertex[index.z];

        RadianceCellScatterPRD prd = loadRadianceCellScatterPRD();
        float distanceToIntersection = (((intersectionWorldPos.x - prd.rayOrigin.x) * (intersectionWorldPos.x - prd.rayOrigin.x)) + ((intersectionWorldPos.y - prd.rayOrigin.y) * (intersectionWorldPos.y - prd.rayOrigin.y)) + ((intersectionWorldPos.z - prd.rayOrigin.z) * (intersectionWorldPos.z - prd.rayOrigin.z)));

        prd.distanceToClosestIntersection = distanceToIntersection;
        storeRadianceCellScatterPRD(prd);
    }

    extern "C" __global__ void __anyhit__radiance__cell__scattering__scene() {
        // Do nothing
    }

    extern "C" __global__ void __miss__radiance__cell__scattering()
    {
        // Do nothing
    }

    extern "C" __global__ void __raygen__renderFrame__cell__scattering()
    {
        const int uvIndex = optixGetLaunchIndex().x;

        // Take different seed for each radiance cell face
        unsigned int seed = tea<4>(uvIndex, optixLaunchParams.nonEmptyCellIndex);

        // Get UV world position for this shader pass
        const int uvInsideOffset = optixLaunchParams.uvsInsideOffsets[optixLaunchParams.nonEmptyCellIndex];
        glm::vec2 uv = optixLaunchParams.uvsInside[uvInsideOffset + uvIndex];
        const int u = int(uv.x * optixLaunchParams.uvWorldPositions.size);
        const int v = int(uv.y * optixLaunchParams.uvWorldPositions.size);

        glm::vec3 UVWorldPos = optixLaunchParams.uvWorldPositions.UVDataBuffer[v * optixLaunchParams.uvWorldPositions.size + u].worldPosition;
        const glm::vec3 UVNormal = optixLaunchParams.uvWorldPositions.UVDataBuffer[v * optixLaunchParams.uvWorldPositions.size + u].worldNormal;

        float3 uvNormal3f = float3{ UVNormal.x, UVNormal.y, UVNormal.z };

        // Center of this radiance cell
        glm::vec3 cellCenter = optixLaunchParams.cellCenter;
        float cellSize = optixLaunchParams.cellSize;
        float stratifyCellWidth = cellSize / optixLaunchParams.stratifyResX;
        float stratifyCellHeight = cellSize / optixLaunchParams.stratifyResY;
        float invCellVolume = 1.0f / (cellSize * cellSize * cellSize);

        float stratifyCellWidthNormalized = 1.0f / optixLaunchParams.stratifyResX;
        float stratifyCellHeightNormalized = 1.0f / optixLaunchParams.stratifyResY;

        // SH weights for this cell
        float SHweights[8][9];
        int amountBasisFunctions = optixLaunchParams.sphericalHarmonicsWeights.amountBasisFunctions;
        int cellOffset = optixLaunchParams.nonEmptyCellIndex * amountBasisFunctions * 8;

        // Load in SH weights from buffer
        for (int sh_i = 0; sh_i < 8; sh_i++)
        {
            for (int basis_f_i = 0; basis_f_i < 9; basis_f_i++)
            {
                SHweights[sh_i][basis_f_i] = optixLaunchParams.sphericalHarmonicsWeights.weights[cellOffset + sh_i * amountBasisFunctions + basis_f_i];
            }
        }

        float3 ogLeft{ cellCenter.x - 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z + 0.5f * cellSize };
        float3 ogRight{ cellCenter.x + 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z - 0.5f * cellSize };
        float3 ogUp{ cellCenter.x - 0.5f * cellSize, cellCenter.y + 0.5f * cellSize, cellCenter.z - 0.5f * cellSize };
        float3 ogDown{ cellCenter.x - 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z + 0.5f * cellSize };
        float3 ogFront{ cellCenter.x - 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z - 0.5f * cellSize };
        float3 ogBack{ cellCenter.x + 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z + 0.5f * cellSize };

        // LEFT, RIGHT, UP, DOWN, FRONT, BACK
        float3 cellNormals[6] = { float3{-1.0f, 0.0f, 0.0f}, float3{1.0f, 0.0f, 0.0f}, float3{0.0f, 1.0f, 0.0f}, float3{0.0f, -1.0f, 0.0f}, float3{0.0f, 0.0f, -1.0f}, float3{0.0f, 0.0f, 1.0f} };
        // Origin, du, dv for each face
        float3 faceOgDuDv[6][3] = { {ogLeft, float3{0.0f, 0.0f, -1.0f}, float3{0.0f, 1.0f, 0.0f} }, {ogRight, float3{0.0f, 0.0f, 1.0f},float3{0.0f, 1.0f, 0.0f} }, {ogUp, float3{1.0f, 0.0f, 0.0f},float3{0.0f, 0.0f, 1.0f} }, {ogDown, float3{1.0f, 0.0f, 0.0f},float3{0.0f, 0.0f, -1.0f}}, {ogFront, float3{1.0f, 0.0f, 0.0f},float3{0.0f, 1.0f, 0.0f} }, {ogBack, float3{-1.0f, 0.0f, 0.0f},float3{0.0f, 1.0f, 0.0f} } };
        // The indices of the SHs that belong to each face, to use while indexing the buffer (L,R,U,D,F,B), (LB, RB, LT, RT)
        int4 cellSHIndices[6] = { int4{4, 0, 6, 2}, int4{1, 5, 3, 7}, int4{2, 3, 6, 7}, int4{4, 5, 0, 1}, int4{0, 1, 2, 3}, int4{5, 4, 7, 6} };

        // Irradiance accumulator
        float totalIrradiance = 0.0f;

        // Loop over cell faces
        for (int face = 0; face < 6; face++)
        {
            // Which SHs of the cell belong to this face
            int4 faceSHIndices = cellSHIndices[face];

            glm::vec3 og = glm::vec3{ faceOgDuDv[face][0].x,faceOgDuDv[face][0].y,faceOgDuDv[face][0].z };
            glm::vec3 du = glm::vec3{ faceOgDuDv[face][1].x,faceOgDuDv[face][1].y,faceOgDuDv[face][1].z };
            glm::vec3 dv = glm::vec3{ faceOgDuDv[face][2].x,faceOgDuDv[face][2].y,faceOgDuDv[face][2].z };

            // Face normal and UV normal need to point in the same direction (hemisphere) for the UV to get contribution from that face
            double cellFaceFacing = dot(uvNormal3f, cellNormals[face]);
            if (cellFaceFacing > 0)
            {
                // For each stratified cell on the face, take samples
                for (int stratifyIndexX = 0; stratifyIndexX < optixLaunchParams.stratifyResX; stratifyIndexX++)
                {
                    for (int stratifyIndexY = 0; stratifyIndexY < optixLaunchParams.stratifyResY; stratifyIndexY++)
                    {
                        glm::vec3 stratifyCellOrigin = og + (stratifyIndexX * stratifyCellWidth * du) + (stratifyIndexY * stratifyCellHeight * dv);

                        // Send out a ray for each sample
                        for (int sample = 0; sample < NUM_SAMPLES_PER_STRATIFY_CELL; sample++)
                        {
                            // Take a random sample on the face's stratified cell, this will be the ray origin
                            float2 randomOffset = float2{ rnd(seed), rnd(seed) };
                            glm::vec3 rayOrigin = stratifyCellOrigin + (randomOffset.x * stratifyCellWidth * du) + (randomOffset.y * stratifyCellHeight * dv);

                            // Ray direction (from the sample to the UV texel)
                            glm::vec3 rayDir = rayOrigin - UVWorldPos;

                            // Convert to float3 format
                            float3 rayOrigin3f = float3{ rayOrigin.x, rayOrigin.y, rayOrigin.z };
                            float3 rayDir3f = float3{ rayDir.x, rayDir.y, rayDir.z };

                            // Calculate spherical coordinate representation of ray
                            // (https://en.wikipedia.org/wiki/Spherical_coordinate_system#Cartesian_coordinates)
                            float3 normalizedRayDir = normalize(rayDir3f);
                            double theta = acos(normalizedRayDir.z);
                            int signY = signbit(normalizedRayDir.y) == 0 ? 1 : -1;
                            double phi = signY * acos(normalizedRayDir.x / (sqrtf((normalizedRayDir.x * normalizedRayDir.x) + (normalizedRayDir.y * normalizedRayDir.y))));

                            RadianceCellScatterPRD prd{};
                            prd.rayOrigin = rayOrigin;

                            unsigned int u0, u1, u2, u3;

                            u1 = __float_as_uint(prd.rayOrigin.x);
                            u2 = __float_as_uint(prd.rayOrigin.y);
                            u3 = __float_as_uint(prd.rayOrigin.z);

                            // Trace ray against scene geometry to see if ray is occluded
                            optixTrace(optixLaunchParams.sceneTraversable,
                                rayOrigin3f,
                                rayDir3f,
                                0.f,    // tmin
                                1e20f,  // tmax
                                0.0f,   // rayTime
                                OptixVisibilityMask(255),
                                OPTIX_RAY_FLAG_DISABLE_ANYHIT,      // We only need closest-hit for scene geometry
                                0,  // SBT offset
                                1,  // SBT stride
                                0,  // missSBTIndex
                                u0, u1, u2, u3
                            );

                            prd.distanceToClosestIntersection = u0;
                            float distanceToUV = (((UVWorldPos.x - prd.rayOrigin.x) * (UVWorldPos.x - prd.rayOrigin.x)) + ((UVWorldPos.y - prd.rayOrigin.y) * (UVWorldPos.y - prd.rayOrigin.y)) + ((UVWorldPos.z - prd.rayOrigin.z) * (UVWorldPos.z - prd.rayOrigin.z)));

                            if (distanceToUV < prd.distanceToClosestIntersection)
                            {
                                // We calculate the UV world position's offset within the cell to do
                                // the calculation of the weights for trilinear interpolation
                                glm::vec3 diff = UVWorldPos - og;

                                // Calculate trilinear interpolation weights, see thesis for explanation
                                float weightA = (diff.x * diff.y * diff.z) * invCellVolume;
                                float weightB = ((1.0f - diff.x) * diff.y * diff.z) * invCellVolume;
                                float weightC = (diff.x * (1.0f - diff.y) * diff.z) * invCellVolume;
                                float weightD = ((1.0f - diff.x) * (1.0f * diff.y) * diff.z) * invCellVolume;
                                float weightE = (diff.x * diff.y * (1.0f - diff.z)) * invCellVolume;
                                float weightF = ((1.0f - diff.x) * diff.y * (1.0f - diff.z)) * invCellVolume;
                                float weightG = (diff.x * (1.0f - diff.y) * (1.0f - diff.z)) * invCellVolume;
                                float weightH = ((1.0f - diff.x) * (1.0f - diff.y) * (1.0f - diff.z)) * invCellVolume;

                                // Basis function evaluations
                                float b0 = Y_0_0();
                                float b1 = Y_min1_1(phi, theta);
                                float b2 = Y_0_1(phi, theta);
                                float b3 = Y_1_1(phi, theta);
                                float b4 = Y_min2_2(phi, theta);
                                float b5 = Y_min1_2(phi, theta);
                                float b6 = Y_0_2(phi, theta);
                                float b7 = Y_1_2(phi, theta);
                                float b8 = Y_2_2(phi, theta);

                                // Calculate the outcoming weight to apply to each basis function
                                float w0 = SHweights[0][0] * weightA + SHweights[1][0] * weightB + SHweights[2][0] * weightC + SHweights[3][0] * weightD + SHweights[4][0] * weightE + SHweights[5][0] * weightF + SHweights[6][0] * weightG + SHweights[7][0] * weightF;
                                float w1 = SHweights[0][1] * weightA + SHweights[1][1] * weightB + SHweights[2][1] * weightC + SHweights[3][1] * weightD + SHweights[4][1] * weightE + SHweights[5][1] * weightF + SHweights[6][1] * weightG + SHweights[7][1] * weightF;
                                float w2 = SHweights[0][2] * weightA + SHweights[1][2] * weightB + SHweights[2][2] * weightC + SHweights[3][2] * weightD + SHweights[4][2] * weightE + SHweights[5][2] * weightF + SHweights[6][2] * weightG + SHweights[7][2] * weightF;
                                float w3 = SHweights[0][3] * weightA + SHweights[1][3] * weightB + SHweights[2][3] * weightC + SHweights[3][3] * weightD + SHweights[4][3] * weightE + SHweights[5][3] * weightF + SHweights[6][3] * weightG + SHweights[7][3] * weightF;
                                float w4 = SHweights[0][4] * weightA + SHweights[1][4] * weightB + SHweights[2][4] * weightC + SHweights[3][4] * weightD + SHweights[4][4] * weightE + SHweights[5][4] * weightF + SHweights[6][4] * weightG + SHweights[7][4] * weightF;
                                float w5 = SHweights[0][5] * weightA + SHweights[1][5] * weightB + SHweights[2][5] * weightC + SHweights[3][5] * weightD + SHweights[4][5] * weightE + SHweights[5][5] * weightF + SHweights[6][5] * weightG + SHweights[7][5] * weightF;
                                float w6 = SHweights[0][6] * weightA + SHweights[1][6] * weightB + SHweights[2][6] * weightC + SHweights[3][6] * weightD + SHweights[4][6] * weightE + SHweights[5][6] * weightF + SHweights[6][6] * weightG + SHweights[7][6] * weightF;
                                float w7 = SHweights[0][7] * weightA + SHweights[1][7] * weightB + SHweights[2][7] * weightC + SHweights[3][7] * weightD + SHweights[4][7] * weightE + SHweights[5][7] * weightF + SHweights[6][7] * weightG + SHweights[7][7] * weightF;
                                float w8 = SHweights[0][8] * weightA + SHweights[1][8] * weightB + SHweights[2][8] * weightC + SHweights[3][8] * weightD + SHweights[4][8] * weightE + SHweights[5][8] * weightF + SHweights[6][8] * weightG + SHweights[7][8] * weightF;

                                // SH reconstruction
                                float irradiance = (b0 * w0) + (b1 * w1) + (b2 * w2) + (b3 * w3) + (b4 * w4) + (b5 * w5) + (b6 * w6) + (b7 * w7) + (b8 * w8);
                                float cosContribution = dot(normalize(rayDir3f), uvNormal3f);

                                totalIrradiance += cosContribution * irradiance;
                            }
                        }
                    }
                }
            }
        }

        int numSamples = 6 * NUM_SAMPLES_PER_STRATIFY_CELL * optixLaunchParams.stratifyResX * optixLaunchParams.stratifyResY;
        // printf("Total irradiance: %f\n", totalIrradiance);

        const int r = int(255.99 * totalIrradiance);
        const int g = int(255.99 * totalIrradiance);
        const int b = int(255.99 * totalIrradiance);

        // convert to 32-bit rgba value (we explicitly set alpha to 0xff
        // to make stb_image_write happy ...
        const uint32_t rgba = 0xff000000
            | (r << 0) | (g << 8) | (b << 16);

        optixLaunchParams.currentBounceTexture.colorBuffer[v * optixLaunchParams.uvWorldPositions.size + u] = rgba;
    }
}